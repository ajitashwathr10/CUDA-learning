
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void VectorAdd(const float *A, const float *B, float *C, int num) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < num) C[i] = A[i] + B[i];
    // Condition to check out of bounds
}

int main() {
    int num = 50000;
    size_t size = num * sizeof(float);
    //Hosting CPU vectors
    float *h_A = (float*)malloc(size);
    float *h_B = (float*)malloc(size);
    float *h_C = (float*)malloc(size);
    //Input vectors
    for (int i = 0; i < num; i++) {
        h_A[i] = rand() / (float)RAND_MAX;
        h_B[i] = rand() / (float)RAND_MAX;
    }
    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);
    
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);
    
    // Using 256 threads per block
    int threads = 256;
    int blocks = (num + threads - 1) / threads;
    printf("CUDA kernel launch with %d blocks of %d threads\n", blocks, threads);
    VectorAdd<<<blocks, threads>>>(d_A, d_B, d_C, num);
    // Check for errors (if there)
    hipError_t status = hipGetLastError();
    if (status != hipSuccess) {
        fprintf(stderr, "VectorAdd launch failed: %s\n", hipGetErrorString(status));
        hipFree(d_A);
        hipFree(d_B);
        hipFree(d_C);
        free(h_A);
        free(h_B);
        free(h_C);
        return -1;
    }
    
    status = hipDeviceSynchronize();
    if (status != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize failed: %s\n", hipGetErrorString(status));
        hipFree(d_A);
        hipFree(d_B);
        hipFree(d_C);
        free(h_A);
        free(h_B);
        free(h_C);
        return -1;
    }
    
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);
    printf("Verifying result\n");
    for (int i = 0; i < 5; i++) {
        printf("h_A[%d] = %f, h_B[%d] = %f, h_C[%d] = %f\n", i, h_A[i], i, h_B[i], i, h_C[i]);
    }
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(h_A);
    free(h_B);
    free(h_C);
    return 0;
}
