
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void HelloWorld() {
    printf("Hello World from Thread %d in Block %d!\n", threadIdx.x, blockIdx.x);
}

int main() {
    printf("Hello World (from CPU)!\n");
    
    HelloWorld<<<2, 4>>>();
    //Launching kernel with 2 blocks (with 4 threads each)
    hipDeviceSynchronize();
    //Waiting for GPU to finish
    hipError_t error = hipGetLastError();
    if(error != hipSuccess) {
        fprintf(stderr, "ERROR: %s\n", hipGetErrorString(error));
        return -1;
    }
    printf("Goodbye World (from CPU)!\n"); 
    return 0;
}
